#include <assert.h>
#include <stdio.h>
#include "GOL_AN5D_1024_kernel.hu"
#include <stdio.h>
#include <stdlib.h>

#define TIMESTEP 1024
#define BENCH_RAD 1
#define SIDE 1026
#define CELL_TYPE char

#define DEAD   0
#define ALIVE  1
#define CELL_NEIGHBOURS 8
#define SRAND_VALUE 1985

void main_computation (CELL_TYPE (*grid)[SIDE][SIDE])
{
	CELL_TYPE rule_table[2][CELL_NEIGHBOURS+1] = {
		    {DEAD,DEAD,DEAD,ALIVE, DEAD,DEAD,DEAD,DEAD,DEAD}, // DEAD is current state
		    {DEAD,DEAD,ALIVE,ALIVE, DEAD,DEAD,DEAD,DEAD,DEAD}  // ALIVE is current state
	};

    {
#define cudaCheckReturn(ret) \
  do { \
    hipError_t cudaCheckReturn_e = (ret); \
    if (cudaCheckReturn_e != hipSuccess) { \
      fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(cudaCheckReturn_e)); \
      fflush(stderr); \
    } \
    assert(cudaCheckReturn_e == hipSuccess); \
  } while(0)
#define cudaCheckKernel() \
  do { \
    cudaCheckReturn(hipGetLastError()); \
  } while(0)

      char *dev_grid;
      char *dev_rule_table;
      
      cudaCheckReturn(hipMalloc((void **) &dev_grid, (size_t)(2) * (size_t)(1026) * (size_t)(1026) * sizeof(char)));
      cudaCheckReturn(hipMalloc((void **) &dev_rule_table, (size_t)(2) * (size_t)(9) * sizeof(char)));
      
{
      cudaCheckReturn(hipMemcpy(dev_grid, grid, (size_t)(2) * (size_t)(1026) * (size_t)(1026) * sizeof(char), hipMemcpyHostToDevice));
#ifdef STENCILBENCH
hipDeviceSynchronize();
SB_START_INSTRUMENTS;
#endif
}
{
      cudaCheckReturn(hipMemcpy(dev_rule_table, rule_table, (size_t)(2) * (size_t)(9) * sizeof(char), hipMemcpyHostToDevice));
#ifdef STENCILBENCH
hipDeviceSynchronize();
SB_START_INSTRUMENTS;
#endif
}
    {
#ifndef AN5D_TYPE
#define AN5D_TYPE unsigned
#endif
      const AN5D_TYPE __c0Len = (1023 - 0 + 1);
      const AN5D_TYPE __c0Pad = (0);
      #define __c0 c0
      const AN5D_TYPE __c1Len = (1024 - 1 + 1);
      const AN5D_TYPE __c1Pad = (1);
      #define __c1 c1
      const AN5D_TYPE __c2Len = (1024 - 1 + 1);
      const AN5D_TYPE __c2Pad = (1);
      #define __c2 c2
      const AN5D_TYPE __halo1 = 1;
      const AN5D_TYPE __halo2 = 1;
      AN5D_TYPE c0;
      AN5D_TYPE __side0LenMax;
      {
        const AN5D_TYPE __side0Len = 4;
        const AN5D_TYPE __side1Len = 128;
        const AN5D_TYPE __side2Len = 24;
        const AN5D_TYPE __OlLen1 = (__halo1 * __side0Len);
        const AN5D_TYPE __OlLen2 = (__halo2 * __side0Len);
        const AN5D_TYPE __side1LenOl = (__side1Len + 2 * __OlLen1);
        const AN5D_TYPE __side2LenOl = (__side2Len + 2 * __OlLen2);
        const AN5D_TYPE __blockSize = 1 * __side2LenOl;
        assert((__side1Len >= 2 * __side0Len * __halo1) && (__c1Len % __side1Len == 0 || __c1Len % __side1Len >= 2 * __side0Len * __halo1) && "[AN5D ERROR] Too short stream");
        dim3 k0_dimBlock(__blockSize, 1, 1);
        dim3 k0_dimGrid(1 * ((__c1Len + __side1Len - 1) / __side1Len) * ((__c2Len + __side2Len - 1) / __side2Len), 1, 1);
        AN5D_TYPE __c0Padr = (__c0Len % 2) != (((__c0Len + __side0Len - 1) / __side0Len) % 2) && __c0Len % __side0Len < 2 ? 1 : 0;
        __side0LenMax = __side0Len;
        for (c0 = __c0Pad; c0 < __c0Pad + __c0Len / __side0Len - __c0Padr; c0 += 1)
        {
          kernel0_4<<<k0_dimGrid, k0_dimBlock>>> (dev_grid, dev_rule_table, c0);
        }
      }
      if ((__c0Len % 2) != (((__c0Len + __side0LenMax - 1) / __side0LenMax) % 2))
      {
        if (__c0Len % __side0LenMax == 0)
        {
          {
            const AN5D_TYPE __side0Len = 2;
            const AN5D_TYPE __side1Len = 128;
            const AN5D_TYPE __side2Len = 28;
            const AN5D_TYPE __OlLen1 = (__halo1 * __side0Len);
            const AN5D_TYPE __OlLen2 = (__halo2 * __side0Len);
            const AN5D_TYPE __side1LenOl = (__side1Len + 2 * __OlLen1);
            const AN5D_TYPE __side2LenOl = (__side2Len + 2 * __OlLen2);
            const AN5D_TYPE __blockSize = 1 * __side2LenOl;
            assert((__side1Len >= 2 * __side0Len * __halo1) && (__c1Len % __side1Len == 0 || __c1Len % __side1Len >= 2 * __side0Len * __halo1) && "[AN5D ERROR] Too short stream");
            dim3 k0_dimBlock(__blockSize, 1, 1);
            dim3 k0_dimGrid(1 * ((__c1Len + __side1Len - 1) / __side1Len) * ((__c2Len + __side2Len - 1) / __side2Len), 1, 1);
            kernel0_2<<<k0_dimGrid, k0_dimBlock>>> (dev_grid, dev_rule_table, c0);
          }
          c0 += 1;
          {
            const AN5D_TYPE __side0Len = 2;
            const AN5D_TYPE __side1Len = 128;
            const AN5D_TYPE __side2Len = 28;
            const AN5D_TYPE __OlLen1 = (__halo1 * __side0Len);
            const AN5D_TYPE __OlLen2 = (__halo2 * __side0Len);
            const AN5D_TYPE __side1LenOl = (__side1Len + 2 * __OlLen1);
            const AN5D_TYPE __side2LenOl = (__side2Len + 2 * __OlLen2);
            const AN5D_TYPE __blockSize = 1 * __side2LenOl;
            assert((__side1Len >= 2 * __side0Len * __halo1) && (__c1Len % __side1Len == 0 || __c1Len % __side1Len >= 2 * __side0Len * __halo1) && "[AN5D ERROR] Too short stream");
            dim3 k0_dimBlock(__blockSize, 1, 1);
            dim3 k0_dimGrid(1 * ((__c1Len + __side1Len - 1) / __side1Len) * ((__c2Len + __side2Len - 1) / __side2Len), 1, 1);
            kernel0_2<<<k0_dimGrid, k0_dimBlock>>> (dev_grid, dev_rule_table, c0);
          }
        }
        else if (__c0Len % __side0LenMax == 1)
        {
          {
            const AN5D_TYPE __side0Len = 3;
            const AN5D_TYPE __side1Len = 128;
            const AN5D_TYPE __side2Len = 26;
            const AN5D_TYPE __OlLen1 = (__halo1 * __side0Len);
            const AN5D_TYPE __OlLen2 = (__halo2 * __side0Len);
            const AN5D_TYPE __side1LenOl = (__side1Len + 2 * __OlLen1);
            const AN5D_TYPE __side2LenOl = (__side2Len + 2 * __OlLen2);
            const AN5D_TYPE __blockSize = 1 * __side2LenOl;
            assert((__side1Len >= 2 * __side0Len * __halo1) && (__c1Len % __side1Len == 0 || __c1Len % __side1Len >= 2 * __side0Len * __halo1) && "[AN5D ERROR] Too short stream");
            dim3 k0_dimBlock(__blockSize, 1, 1);
            dim3 k0_dimGrid(1 * ((__c1Len + __side1Len - 1) / __side1Len) * ((__c2Len + __side2Len - 1) / __side2Len), 1, 1);
            kernel0_3<<<k0_dimGrid, k0_dimBlock>>> (dev_grid, dev_rule_table, c0);
          }
          c0 += 1;
          {
            const AN5D_TYPE __side0Len = 1;
            const AN5D_TYPE __side1Len = 128;
            const AN5D_TYPE __side2Len = 30;
            const AN5D_TYPE __OlLen1 = (__halo1 * __side0Len);
            const AN5D_TYPE __OlLen2 = (__halo2 * __side0Len);
            const AN5D_TYPE __side1LenOl = (__side1Len + 2 * __OlLen1);
            const AN5D_TYPE __side2LenOl = (__side2Len + 2 * __OlLen2);
            const AN5D_TYPE __blockSize = 1 * __side2LenOl;
            assert((__side1Len >= 2 * __side0Len * __halo1) && (__c1Len % __side1Len == 0 || __c1Len % __side1Len >= 2 * __side0Len * __halo1) && "[AN5D ERROR] Too short stream");
            dim3 k0_dimBlock(__blockSize, 1, 1);
            dim3 k0_dimGrid(1 * ((__c1Len + __side1Len - 1) / __side1Len) * ((__c2Len + __side2Len - 1) / __side2Len), 1, 1);
            kernel0_1<<<k0_dimGrid, k0_dimBlock>>> (dev_grid, dev_rule_table, c0);
          }
          c0 += 1;
          {
            const AN5D_TYPE __side0Len = 1;
            const AN5D_TYPE __side1Len = 128;
            const AN5D_TYPE __side2Len = 30;
            const AN5D_TYPE __OlLen1 = (__halo1 * __side0Len);
            const AN5D_TYPE __OlLen2 = (__halo2 * __side0Len);
            const AN5D_TYPE __side1LenOl = (__side1Len + 2 * __OlLen1);
            const AN5D_TYPE __side2LenOl = (__side2Len + 2 * __OlLen2);
            const AN5D_TYPE __blockSize = 1 * __side2LenOl;
            assert((__side1Len >= 2 * __side0Len * __halo1) && (__c1Len % __side1Len == 0 || __c1Len % __side1Len >= 2 * __side0Len * __halo1) && "[AN5D ERROR] Too short stream");
            dim3 k0_dimBlock(__blockSize, 1, 1);
            dim3 k0_dimGrid(1 * ((__c1Len + __side1Len - 1) / __side1Len) * ((__c2Len + __side2Len - 1) / __side2Len), 1, 1);
            kernel0_1<<<k0_dimGrid, k0_dimBlock>>> (dev_grid, dev_rule_table, c0);
          }
        }
        else if (__c0Len % __side0LenMax == 2)
        {
          {
            const AN5D_TYPE __side0Len = 1;
            const AN5D_TYPE __side1Len = 128;
            const AN5D_TYPE __side2Len = 30;
            const AN5D_TYPE __OlLen1 = (__halo1 * __side0Len);
            const AN5D_TYPE __OlLen2 = (__halo2 * __side0Len);
            const AN5D_TYPE __side1LenOl = (__side1Len + 2 * __OlLen1);
            const AN5D_TYPE __side2LenOl = (__side2Len + 2 * __OlLen2);
            const AN5D_TYPE __blockSize = 1 * __side2LenOl;
            assert((__side1Len >= 2 * __side0Len * __halo1) && (__c1Len % __side1Len == 0 || __c1Len % __side1Len >= 2 * __side0Len * __halo1) && "[AN5D ERROR] Too short stream");
            dim3 k0_dimBlock(__blockSize, 1, 1);
            dim3 k0_dimGrid(1 * ((__c1Len + __side1Len - 1) / __side1Len) * ((__c2Len + __side2Len - 1) / __side2Len), 1, 1);
            kernel0_1<<<k0_dimGrid, k0_dimBlock>>> (dev_grid, dev_rule_table, c0);
          }
          c0 += 1;
          {
            const AN5D_TYPE __side0Len = 1;
            const AN5D_TYPE __side1Len = 128;
            const AN5D_TYPE __side2Len = 30;
            const AN5D_TYPE __OlLen1 = (__halo1 * __side0Len);
            const AN5D_TYPE __OlLen2 = (__halo2 * __side0Len);
            const AN5D_TYPE __side1LenOl = (__side1Len + 2 * __OlLen1);
            const AN5D_TYPE __side2LenOl = (__side2Len + 2 * __OlLen2);
            const AN5D_TYPE __blockSize = 1 * __side2LenOl;
            assert((__side1Len >= 2 * __side0Len * __halo1) && (__c1Len % __side1Len == 0 || __c1Len % __side1Len >= 2 * __side0Len * __halo1) && "[AN5D ERROR] Too short stream");
            dim3 k0_dimBlock(__blockSize, 1, 1);
            dim3 k0_dimGrid(1 * ((__c1Len + __side1Len - 1) / __side1Len) * ((__c2Len + __side2Len - 1) / __side2Len), 1, 1);
            kernel0_1<<<k0_dimGrid, k0_dimBlock>>> (dev_grid, dev_rule_table, c0);
          }
        }
        else if (__c0Len % __side0LenMax == 3)
        {
          {
            const AN5D_TYPE __side0Len = 2;
            const AN5D_TYPE __side1Len = 128;
            const AN5D_TYPE __side2Len = 28;
            const AN5D_TYPE __OlLen1 = (__halo1 * __side0Len);
            const AN5D_TYPE __OlLen2 = (__halo2 * __side0Len);
            const AN5D_TYPE __side1LenOl = (__side1Len + 2 * __OlLen1);
            const AN5D_TYPE __side2LenOl = (__side2Len + 2 * __OlLen2);
            const AN5D_TYPE __blockSize = 1 * __side2LenOl;
            assert((__side1Len >= 2 * __side0Len * __halo1) && (__c1Len % __side1Len == 0 || __c1Len % __side1Len >= 2 * __side0Len * __halo1) && "[AN5D ERROR] Too short stream");
            dim3 k0_dimBlock(__blockSize, 1, 1);
            dim3 k0_dimGrid(1 * ((__c1Len + __side1Len - 1) / __side1Len) * ((__c2Len + __side2Len - 1) / __side2Len), 1, 1);
            kernel0_2<<<k0_dimGrid, k0_dimBlock>>> (dev_grid, dev_rule_table, c0);
          }
          c0 += 1;
          {
            const AN5D_TYPE __side0Len = 1;
            const AN5D_TYPE __side1Len = 128;
            const AN5D_TYPE __side2Len = 30;
            const AN5D_TYPE __OlLen1 = (__halo1 * __side0Len);
            const AN5D_TYPE __OlLen2 = (__halo2 * __side0Len);
            const AN5D_TYPE __side1LenOl = (__side1Len + 2 * __OlLen1);
            const AN5D_TYPE __side2LenOl = (__side2Len + 2 * __OlLen2);
            const AN5D_TYPE __blockSize = 1 * __side2LenOl;
            assert((__side1Len >= 2 * __side0Len * __halo1) && (__c1Len % __side1Len == 0 || __c1Len % __side1Len >= 2 * __side0Len * __halo1) && "[AN5D ERROR] Too short stream");
            dim3 k0_dimBlock(__blockSize, 1, 1);
            dim3 k0_dimGrid(1 * ((__c1Len + __side1Len - 1) / __side1Len) * ((__c2Len + __side2Len - 1) / __side2Len), 1, 1);
            kernel0_1<<<k0_dimGrid, k0_dimBlock>>> (dev_grid, dev_rule_table, c0);
          }
        }
      }
      else if (__c0Len % __side0LenMax)
      {
        if (__c0Len % __side0LenMax == 1)
        {
          const AN5D_TYPE __side0Len = 1;
          const AN5D_TYPE __side1Len = 128;
          const AN5D_TYPE __side2Len = 30;
          const AN5D_TYPE __OlLen1 = (__halo1 * __side0Len);
          const AN5D_TYPE __OlLen2 = (__halo2 * __side0Len);
          const AN5D_TYPE __side1LenOl = (__side1Len + 2 * __OlLen1);
          const AN5D_TYPE __side2LenOl = (__side2Len + 2 * __OlLen2);
          const AN5D_TYPE __blockSize = 1 * __side2LenOl;
          assert((__side1Len >= 2 * __side0Len * __halo1) && (__c1Len % __side1Len == 0 || __c1Len % __side1Len >= 2 * __side0Len * __halo1) && "[AN5D ERROR] Too short stream");
          dim3 k0_dimBlock(__blockSize, 1, 1);
          dim3 k0_dimGrid(1 * ((__c1Len + __side1Len - 1) / __side1Len) * ((__c2Len + __side2Len - 1) / __side2Len), 1, 1);
          kernel0_1<<<k0_dimGrid, k0_dimBlock>>> (dev_grid, dev_rule_table, c0);
        }
        else if (__c0Len % __side0LenMax == 2)
        {
          const AN5D_TYPE __side0Len = 2;
          const AN5D_TYPE __side1Len = 128;
          const AN5D_TYPE __side2Len = 28;
          const AN5D_TYPE __OlLen1 = (__halo1 * __side0Len);
          const AN5D_TYPE __OlLen2 = (__halo2 * __side0Len);
          const AN5D_TYPE __side1LenOl = (__side1Len + 2 * __OlLen1);
          const AN5D_TYPE __side2LenOl = (__side2Len + 2 * __OlLen2);
          const AN5D_TYPE __blockSize = 1 * __side2LenOl;
          assert((__side1Len >= 2 * __side0Len * __halo1) && (__c1Len % __side1Len == 0 || __c1Len % __side1Len >= 2 * __side0Len * __halo1) && "[AN5D ERROR] Too short stream");
          dim3 k0_dimBlock(__blockSize, 1, 1);
          dim3 k0_dimGrid(1 * ((__c1Len + __side1Len - 1) / __side1Len) * ((__c2Len + __side2Len - 1) / __side2Len), 1, 1);
          kernel0_2<<<k0_dimGrid, k0_dimBlock>>> (dev_grid, dev_rule_table, c0);
        }
        else if (__c0Len % __side0LenMax == 3)
        {
          const AN5D_TYPE __side0Len = 3;
          const AN5D_TYPE __side1Len = 128;
          const AN5D_TYPE __side2Len = 26;
          const AN5D_TYPE __OlLen1 = (__halo1 * __side0Len);
          const AN5D_TYPE __OlLen2 = (__halo2 * __side0Len);
          const AN5D_TYPE __side1LenOl = (__side1Len + 2 * __OlLen1);
          const AN5D_TYPE __side2LenOl = (__side2Len + 2 * __OlLen2);
          const AN5D_TYPE __blockSize = 1 * __side2LenOl;
          assert((__side1Len >= 2 * __side0Len * __halo1) && (__c1Len % __side1Len == 0 || __c1Len % __side1Len >= 2 * __side0Len * __halo1) && "[AN5D ERROR] Too short stream");
          dim3 k0_dimBlock(__blockSize, 1, 1);
          dim3 k0_dimGrid(1 * ((__c1Len + __side1Len - 1) / __side1Len) * ((__c2Len + __side2Len - 1) / __side2Len), 1, 1);
          kernel0_3<<<k0_dimGrid, k0_dimBlock>>> (dev_grid, dev_rule_table, c0);
        }
      }
    }
    cudaCheckKernel();
{
#ifdef STENCILBENCH
hipDeviceSynchronize();
SB_STOP_INSTRUMENTS;
#endif
      cudaCheckReturn(hipMemcpy(grid, dev_grid, (size_t)(2) * (size_t)(1026) * (size_t)(1026) * sizeof(char), hipMemcpyDeviceToHost));
}
      cudaCheckReturn(hipFree(dev_grid));
      cudaCheckReturn(hipFree(dev_rule_table));
    }
}

/**/

void init_grid (CELL_TYPE (*grid)[SIDE][SIDE]) {
    srand(SRAND_VALUE);
    for(int i = 1; i<SIDE-1; i++) {
        for(int j = 1; j<SIDE-1; j++) {
            grid[0][i][j] = (CELL_TYPE) (rand() % 2);
        }
    }

}



long int print_total_alive (int ca, CELL_TYPE (*grid)[SIDE][SIDE]) {
	int i,j;
	long int total = 0;
	for (i = 1; i<SIDE-1; i++) {
		for (j = 1; j<SIDE-1; j++) {
			total += grid[ca][i][j];
		}
	}
	//printf("Total Alive: %d\n", total);
	return total;
}



int main(int argc, char* argv[])
{
    // Define variables
    int i,j;
    long int total = 0;
    CELL_TYPE (*grid)[SIDE][SIDE];
 

    // Allocate grid
    grid = (CELL_TYPE (*)[SIDE][SIDE]) malloc (2*sizeof(CELL_TYPE)*(SIDE)*(SIDE));  // grid[0][][] --> current CA state
                                                                                // grid[1][][] --> next CA state
  
    // Assign initial population randomly
    init_grid (grid);

 
    // Main GOL game loop
    main_computation (grid);


    // Sum up alive cells and print results
    printf("Total Alive CA 0: %ld\n", print_total_alive (0, grid));
    //printf("Total Alive CA 1: %ld\n", print_total_alive (1, grid));
 

    // Release memory
    free(grid);

}
/**/


// 256 Result in console: "Total Alive: 3281"
// 512 Result in console: "Total Alive: 11072"
// 1024 Result in console: "Total Alive: 45224"
// 2048 Result in console: "Total Alive: 182485"
// 4096 Result in console: "Total Alive: 724393"
// 8192 Result in console: "Total Alive: 2896683"

